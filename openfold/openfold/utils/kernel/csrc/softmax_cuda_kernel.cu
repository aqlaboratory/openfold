#include "hip/hip_runtime.h"
// Copyright 2021 AlQuraishi Laboratory
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// modified from fastfold/model/fastnn/kernel/cuda_native/csrc/softmax_cuda_kernel.cu

#include <hip/hip_math_constants.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include <iostream>

#include "ATen/ATen.h"
#include "ATen/cuda/HIPContext.h"
#include "compat.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)

__inline__ __device__ float WarpAllReduceMax(float val) {
    for (int mask = 1; mask < 32; mask *= 2) {
        val = max(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

__inline__ __device__ float WarpAllReduceSum(float val) {
    for (int mask = 1; mask < 32; mask *= 2) {
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}


template<typename T>
__global__ void attn_softmax_inplace_(
    T *input, 
    long long rows, int cols
) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)(blockIdx.x * 4 + threadidx_x);
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        T *row_input = input + row_offset * cols;
        T *row_output = row_input;

        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            int idx = lane_id * cols_per_thread + i;
            buf[i] = static_cast<float>(row_input[idx]);
        }

        float thread_max = -1 * HIP_INF_F;
        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_max = max(thread_max, buf[i]);
        }

        float warp_max = WarpAllReduceMax(thread_max);

        float thread_sum = 0.f;
        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            buf[i] = __expf(buf[i] - warp_max);
            thread_sum += buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);
        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            row_output[lane_id * cols_per_thread + i] =
                static_cast<T>(__fdividef(buf[i], warp_sum));
        }
    }
}


void attn_softmax_inplace_forward_(
    at::Tensor input, 
    long long rows, int cols
) {
    CHECK_INPUT(input);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(input));

    int grid = (rows + 3) / 4;
    dim3 block(128);

    if (input.dtype() == torch::kFloat32) {
        attn_softmax_inplace_<float><<<grid, block>>>(
            (float *)input.data_ptr(),
            rows, cols
        );
    } 
    else {
        attn_softmax_inplace_<at::BFloat16><<<grid, block>>>(
            (at::BFloat16 *)input.data_ptr(), 
            rows, cols
        );
    }
}


template<typename T>
__global__ void attn_softmax_inplace_grad_(
    T *output,
    T *d_ov,
    T *values,
    long long rows, 
    int cols_output,
    int cols_values
) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)(blockIdx.x * 4 + threadidx_x);
    int cols_per_thread = (cols_output + 31) / 32;
    int cols_this_thread = cols_per_thread;
    int rows_values = cols_output;
    // values are set to the beginning of the current 
    // rows_values x cols_values leaf matrix
    long long value_row_offset = row_offset - row_offset % rows_values;
    int last_y = (cols_output / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols_output - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float y_buf[32];
    float dy_buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        T *row_output = output + row_offset * cols_output;
        T *row_d_ov = d_ov + row_offset * cols_values;
        T *row_values = values + value_row_offset * cols_values;

        float thread_max = -1 * HIP_INF_F;

        // Compute a chunk of the output gradient on the fly
        int value_row_idx = 0;
        int value_idx = 0;
        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            T sum = 0.;
            #pragma unroll
            for (int j = 0; j < cols_values; j++) {
                value_row_idx = ((lane_id * cols_per_thread) + i);
                value_idx = value_row_idx * cols_values + j;
                sum += row_d_ov[j] * row_values[value_idx];
            }
            dy_buf[i] = static_cast<float>(sum);
        }

        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            y_buf[i] = static_cast<float>(row_output[lane_id * cols_per_thread + i]);
        }

        float thread_sum = 0.;

        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_sum += y_buf[i] * dy_buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);

        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            row_output[lane_id * cols_per_thread + i] = static_cast<T>(
                    (dy_buf[i] - warp_sum) * y_buf[i]
            );
        }
    }
}


void attn_softmax_inplace_backward_(
    at::Tensor output,
    at::Tensor d_ov, 
    at::Tensor values,
    long long rows, 
    int cols_output,
    int cols_values
) {
    CHECK_INPUT(output);
    CHECK_INPUT(d_ov);
    CHECK_INPUT(values);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(output));

    int grid = (rows + 3) / 4;
    dim3 block(128);

    if (output.dtype() == torch::kFloat32) {
        attn_softmax_inplace_grad_<float><<<grid, block>>>(
            (float *)output.data_ptr(),
            (float *)d_ov.data_ptr(), 
            (float *)values.data_ptr(),
            rows, cols_output, cols_values
        );
    } else {
        attn_softmax_inplace_grad_<at::BFloat16><<<grid, block>>>(
            (at::BFloat16 *)output.data_ptr(),
            (at::BFloat16 *)d_ov.data_ptr(), 
            (at::BFloat16 *)values.data_ptr(),
            rows, cols_output, cols_values
        );
    }
}
